#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#define _USE_MATH_DEFINES

#include <math.h>
#include <stdio.h>
#include <vector>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

#define BLK_WIDTH 32
#define BLK_HEIGHT 32
#define BLOCKSIZE (BLK_HEIGHT * BLK_WIDTH)
#define SCAN_BLOCK_DIM BLOCKSIZE

#include "exclusiveScan.cu_inl"
#include "circleBoxTest.cu_inl"

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans)  cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
#else
#define cudaCheckError(ans) ans
#endif

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

// This stores the global constants
struct GlobalConstants {

    SceneName sceneName;

    int numberOfCircles;

    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// Read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// Color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// Include parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
// 
// Update positions of fireworks
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = M_PI;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update 
        return;
    }

    // Determine the firework center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // Update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // Firework sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // Compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // Compute distance from fire-work 
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position 
        // Random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // Travel scaled unit length 
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis   
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numberOfCircles) 
        return; 

    float* radius = cuConstRendererParams.radius; 

    float cutOff = 0.5f;
    // Place circle back in center after reaching threshold radisus 
    if (radius[index] > cutOff) { 
        radius[index] = 0.02f; 
    } else { 
        radius[index] += 0.01f; 
    }   
}   


// kernelAdvanceBouncingBalls
// 
// Update the position of the balls
__global__ void kernelAdvanceBouncingBalls() {
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x; 
   
    if (index >= cuConstRendererParams.numberOfCircles) 
        return; 

    float* velocity = cuConstRendererParams.velocity; 
    float* position = cuConstRendererParams.position; 

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition 
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball 
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball 
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// Move the snowflake animation forward one time step.  Update circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numberOfCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // Load from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // Hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // Add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // Drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // Update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // Update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // If the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // Restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // Store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}


// shadePixel -- (CUDA device code)
//
// Given a pixel and a circle, determine the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__
void shadePixel(float2 pixelCenter, float3 p, float4* imagePtr,
                int circleIndex, float rad, SceneName sname) { 
    // Check if point lies within the circle
    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;
    float maxDist = rad * rad;
    if (pixelDist > maxDist) {
        return;
    }

    float3 rgb;
    float alpha;
    // Compute the shading value

    // Suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks, etc., to implement the conditional.  It
    // would be wise to perform this logic outside of the loops in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).

    // NOTE: Get the color and alpha depending on the scene
    if (sname == SNOWFLAKES || sname == SNOWFLAKES_SINGLE_FRAME) {
        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f - p.z);
        // kCircleMaxAlpha * clamped value
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f);
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);
    } else {
        // Simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;
    // Color update
    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;
    *imagePtr = newColor;
}

/**
 * Index checker
 *
 */
__device__  __inline__
void indexChecker(uint *input, int *output, int linearIndex, int circleIndex)
{
    if (input[linearIndex + 1] == 1 + input[linearIndex]) {
        output[input[linearIndex]] = circleIndex;
    }
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__
void kernelRenderCircles(float invWidth, float invHeight, int factor,
                         short imageWidth, short imageHeight, int numberOfCircles) {
    __shared__ uint SumInput[BLOCKSIZE];
    __shared__ uint SumOutput[BLOCKSIZE];
    __shared__ uint SumScratch[2 * BLOCKSIZE];
    __shared__ int circleArr[BLOCKSIZE];

    SceneName sname = cuConstRendererParams.sceneName;
    // Starting index of the block in the image
    int blockMinX = blockIdx.x * blockDim.x;
    int blockMaxX = blockMinX + blockDim.x;
    int blockMinY = blockIdx.y * blockDim.y;
    int blockMaxY = blockMinY + blockDim.y;

    int linearIdx = threadIdx.y * blockDim.y + threadIdx.x;  // 0 - BLOCKSIZE
    int circleIndex = factor + linearIdx;  // global circle index

    // Update block boundaries for blocks at the edges of the image
    blockMaxX = min(blockMaxX, imageWidth);
    blockMaxY = min(blockMaxY, imageHeight);

    // If circle exists
    if (circleIndex < numberOfCircles) {
        // Read position and radius (normalized)
        int index3 = 3 * circleIndex;
        float r = cuConstRendererParams.radius[circleIndex];
        float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
        int overlap = circleInBoxConservative(p.x, p.y, r,
            (static_cast<float>(blockMinX) + 0.5f) * invWidth,
            (static_cast<float>(blockMaxX) + 0.5f) * invWidth,
            (static_cast<float>(blockMaxY) + 0.5f) * invHeight,
            (static_cast<float>(blockMinY) + 0.5f) * invHeight);
        SumInput[linearIdx] = overlap;
    } else {
        SumInput[linearIdx] = 0;
    }

    __syncthreads();  // ensure that all threads have contributed to the mask
    sharedMemExclusiveScan(linearIdx, SumInput, SumOutput, SumScratch, BLOCKSIZE);
    __syncthreads();  // ensure that prefix-sum was stored in SumOutput
    int mycount = SumOutput[BLOCKSIZE - 1];  // WARNING: all threads will be accessing the same location
    if (SumInput[BLOCKSIZE - 1] == 1) {
        mycount += 1;
    }
    indexChecker(SumOutput, circleArr, linearIdx, circleIndex);
    // Check the last index; it is not checked by exclusive scan
    if (linearIdx == BLOCKSIZE - 1 && SumInput[linearIdx] == 1) {
        circleArr[mycount-1] = circleIndex;
    }
    __syncthreads();  // ensure that all the contributing cirles were added
    // Index of the thread in the image
    int threadX = blockMinX + threadIdx.x;  // Pixel x
    int threadY = blockMinY + threadIdx.y;  // Pixel y
    // If pixel is out of the image
    if (threadX >= blockMaxX || threadY >= blockMaxY) {
        return;
    }

    float posx = invWidth * (static_cast<float>(threadX) + 0.5f);
    float posy = invHeight * (static_cast<float>(threadY) + 0.5f);
    float4 imgInfo = *(float4*)(&cuConstRendererParams.imageData[
        4 * (threadY * imageWidth + threadX)]);
    float2 pixelCenterNorm = make_float2(posx, posy);
    
    // Sequentially update for circles
    for (int j = 0; j < mycount; j++) {
        int circleId = circleArr[j];
        float3 p_ = *(float3*)(&cuConstRendererParams.position[circleId * 3]);
        float rad_ = cuConstRendererParams.radius[circleId];
        shadePixel(pixelCenterNorm, p_, &imgInfo, circleId, rad_, sname);
    }

    *(float4*)(&cuConstRendererParams.imageData[
        4 * (threadY * imageWidth + threadX)]) = imgInfo;
}


////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numberOfCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image*
CudaRenderer::getImage() {

    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numberOfCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce RTX 2080") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA RTX 2080.\n");
        printf("---------------------------------------------------------\n");
    }
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numberOfCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numberOfCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numberOfCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numberOfCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numberOfCircles = numberOfCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // Also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // Copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {
    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numberOfCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) { 
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>(); 
    }
    hipDeviceSynchronize();
}


void
CudaRenderer::render() {
    // 256 threads per block is a healthy number
    short imageWidth = image->width;
    short imageHeight = image->height;
    int cols = (imageWidth + BLK_WIDTH - 1) / BLK_WIDTH;
    int rows = (imageHeight + BLK_HEIGHT - 1) / BLK_HEIGHT;

    // Block config
    dim3 blockDim(BLK_WIDTH, BLK_HEIGHT, 1);
    dim3 gridDim(cols, rows);
    int n = (numberOfCircles + BLOCKSIZE - 1) / BLOCKSIZE;
    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;
    
    for (int i = 0; i < n; i++) {
        kernelRenderCircles<<<gridDim, blockDim>>>(invWidth, invHeight,
                                                   i * BLOCKSIZE, imageWidth,
                                                   imageHeight, numberOfCircles);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
}
