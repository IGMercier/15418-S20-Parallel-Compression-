#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../include/config_cuda.hh"

__constant__ int cudaQuantArr[WINDOW_Y * WINDOW_X];
uint8_t *cudaImg;


__device__
void discreteCosTransformCuda(const int *grayData, float *patchDCT, const int &linearIdx) {
    // Put these in the read only memory.
    float cos1, cos2, temp;
    // Useful constansts
    const float term1 = M_PI / (float)WINDOW_X;
    const float term2 = M_PI / (float)WINDOW_Y;
    const float one_by_root_2 = 1.0 / sqrtf(2);
    const float one_by_root_2N = 1.0 / sqrtf(2 * WINDOW_X);

    int x, y;
    temp = 0.0;
    for (y = 0; y < WINDOW_Y; y++) {
        for (x = 0; x < WINDOW_X; x++) {
            // These cosine values can be pre-stored for a (WINDOW_X, WINDOW_Y)
            // matrix
            // TODO: Verify the order of x and y
            cos1 = cosf(term1 * (y + 0.5) * threadIdx.y);
            cos2 = cosf(term2 * (x + 0.5) * threadIdx.x);
            temp += grayData[y * blockDim.x + x] * cos1 * cos2;
        }
    }

    temp *= one_by_root_2N;
    if (threadIdx.y > 0) {
        temp *= one_by_root_2;
    }

    if (threadIdx.x > 0) {
        temp *= one_by_root_2;
    }

    // TODO: ensure that u + offset < i + pixel and < n
    patchDCT[linearIdx] = (int)temp;
}


__device__ __inline__
void quantizeCuda(const float *patchDCT, int *quantData, const int &linearIdx) {
    // TODO: Use a single array for the purpose of quantizing
    quantData[linearIdx] = (int)roundf((float)patchDCT[linearIdx]
                                       / cudaQuantArr[linearIdx]);
}


__device__ __inline__
void dequantizeCuda(const int *quantData, int *dequantData, const int &linearIdx) {
    // TODO: Use a single array for the purpose of quantizing and dequantizing
    dequantData[linearIdx] = quantData[linearIdx] * cudaQuantArr[linearIdx];
}


__device__
void invDiscreteCosTransformCuda(const int *dequantData, int *patchInverseDCT, int offset, const int &linearIdx) {
    int x, y;
    float cos1, cos2, temp;
    // Useful constants.
    const float term1 = M_PI / (float)WINDOW_X;
    const float term2 = M_PI / (float)WINDOW_Y;
    const float term3 = 2. / (float)WINDOW_X;
    const float term4 = 2. / (float)WINDOW_Y;
    // 1st value
    temp = 1/4. * (float)dequantData[0 * blockDim.x + 0];
    // First column values
    for (y = 1; y < WINDOW_Y; y++) {
        temp += 1/2. * (float)dequantData[y * blockDim.x + 0];
    }
    // First row values
    for (x = 1; x < WINDOW_X; x++) {
        temp += 1/2. * (float)dequantData[0 * blockDim.x + x];
    }

    // if (offset == 202944) {
    //     printf("Partial INV DCT: %f\n", temp);
    // }

    for (y = 1; y < WINDOW_Y; y++) {
        for (x = 1; x < WINDOW_X; x++) {
            cos1 = cosf(term1 * (y + 0.5) * threadIdx.y);
            cos2 = cosf(term2 * (x + 0.5) * threadIdx.x);
            temp += (float)dequantData[y * blockDim.x + x] * cos1 * cos2;
        }
    }

    // if (offset == 202944) {
    //     printf("Partial INV DCT: %d\n", temp);
    // }

    patchInverseDCT[linearIdx] = temp * term3 * term4;
    // if (offset == 202944) {
    //     printf("Partial INV DCT: %d\n", patchInverseDCT[threadIdx.y * blockDim.x + threadIdx.x]);
    // }
}


__device__ __inline__
int getOffset(int width, int i, int j) {
    /**
     *  width: image width
     *  i: pixel row
     *  j: pixel column
     */
    return (i * width + j) * NUM_CHANNELS;
}


__global__
void compressCuda(uint8_t *cudaImg, int width, int height) {
    // ASSUMPTION: 8 x 8 block
    // TODO: Can save memory by using only 2 arrays
    const int num = BLOCKSIZE;
    __shared__ int grayData[num];
    __shared__ float patchDCT[num];
    __shared__ int quantData[num];
    __shared__ int dequantData[num];
    __shared__ int patchInverseDCT[num];

    int add_rows = (PIXEL - (height % PIXEL) != PIXEL ? PIXEL - (height % PIXEL) : 0);
    int add_columns = (PIXEL - (width % PIXEL) != PIXEL ? PIXEL - (width % PIXEL) : 0);

    // padded dimensions to make multiples of patch size
    int _height = height + add_rows;
    int _width = width + add_columns;

    int blockMinX = blockIdx.x * blockDim.x;
    int blockMaxX = blockMinX + blockDim.x;
    int blockMinY = blockIdx.y * blockDim.y;
    int blockMaxY = blockMinY + blockDim.y;

    blockMaxX = min(blockMaxX, _width);
    blockMaxY = min(blockMaxY, _height);

    int pixelX = blockMinX + threadIdx.x;
    int pixelY = blockMinY + threadIdx.y;

    int linearIdx = threadIdx.y * blockDim.x + threadIdx.x;
    int offset = getOffset(width, pixelY, pixelX);

    // Write grayscale data in `grayData` along with zero padding
    if (pixelX < width && pixelY < height) {
        uint8_t *bgrPixel = (uint8_t *) &cudaImg[offset];
        grayData[linearIdx] = (bgrPixel[0] + bgrPixel[1] + bgrPixel[2]) / 3.f;
    } else if (pixelX >= width) {
        grayData[linearIdx] = 0;
    } else if (pixelY >= height) {
        grayData[linearIdx] = 0;
    }

    __syncthreads();
    discreteCosTransformCuda(grayData, patchDCT, linearIdx);
    quantizeCuda(patchDCT, quantData, linearIdx);
    dequantizeCuda(quantData, dequantData, linearIdx);
    __syncthreads();
    invDiscreteCosTransformCuda(dequantData, patchInverseDCT, offset, linearIdx);
    __syncthreads();

    if (pixelX >= width || pixelY >= height) {
        return;
    }

    cudaImg[offset + 0] = patchInverseDCT[linearIdx];
    cudaImg[offset + 1] = patchInverseDCT[linearIdx];
    cudaImg[offset + 2] = patchInverseDCT[linearIdx];
}


void cudaSetup(uint8_t *img, int width, int height) {
    size_t num = NUM_CHANNELS * width * height;
    hipMalloc(&cudaImg, sizeof(uint8_t) * num);
    hipMemcpy(cudaImg, img, sizeof(uint8_t) * num, hipMemcpyHostToDevice);

    int quantArr[WINDOW_Y * WINDOW_X] = {16, 11, 12, 14, 12, 10, 16, 14,
                                         13, 14, 18, 17, 16, 19, 24, 40,
                                         26, 24, 22, 22, 24, 49, 35, 37,
                                         29, 40, 58, 51, 61, 60, 57, 51,
                                         56, 55, 64, 72, 92, 78, 64, 68,
                                         87, 69, 55, 56, 80, 109, 81, 87,
                                         95, 98, 103, 104, 103, 62, 77, 113,
                                         121, 112, 100, 120, 92, 101, 103, 99
                                         };
    hipMemcpyToSymbol(HIP_SYMBOL(cudaQuantArr), &quantArr, sizeof(int) * WINDOW_X * WINDOW_Y);

    // TODO: Number of rows and cols should be based on the padded dimensions.
    int rows = (height + BLK_HEIGHT - 1) / BLK_HEIGHT;
    int cols = (width + BLK_WIDTH - 1) / BLK_WIDTH;
    dim3 blockDim(BLK_WIDTH, BLK_HEIGHT);
    dim3 gridDim(cols, rows);

    compressCuda<<<gridDim, blockDim>>>(cudaImg, width, height);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));
}


void cudaFinish(uint8_t *img, int width, int height) {
    size_t num = NUM_CHANNELS * width * height;
    hipMemcpy(img, cudaImg, sizeof(uint8_t) * num, hipMemcpyDeviceToHost);
    hipFree(cudaImg);
}
